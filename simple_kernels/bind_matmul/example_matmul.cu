#include "hip/hip_runtime.h"
#include "kittens.cuh"
#include "pyutils/pyutils.cuh"
using namespace kittens;

#define NUM_THREADS (kittens::WARP_THREADS) // use 1 warp

#define _row 16
#define _col 32

// input is bfloat16, output is float32
// NEED TO CHECK IF THIS IS CORRECT
using x_gl  = gl<bf16, -1, -1, -1, -1, st_bf<_row, _col>>;
using o_gl  = gl<float, -1, -1, -1, -1, st_fl<_row, _row>>;

struct micro_globals {
    x_gl x;
    o_gl o;
    dim3 grid()  { return dim3(o.batch, o.depth, o.rows); }
    dim3 block() { return dim3(o.cols); }
    size_t dynamic_shared_memory() { return 224000; } // I added this but kinda sus
    
};

// make this number of threads in a thread block
__global__ __launch_bounds__(NUM_THREADS, 1)
void micro_tk(const __grid_constant__ micro_globals g) {
    // shared memory
    extern __shared__ alignment_dummy __shm[];
    shared_allocator al((int*)&__shm[0]);
    st_bf<_row, _col> (&x_s) = al.allocate<st_bf<_row, _col>>(); // bf16 tiles
    st_fl<_row, _row> (&o_s) = al.allocate<st_fl<_row, _row>>(); // float tiles

    // register memory
    rt_bf<_row, _col> x_reg; // bf16 register
    rt_fl <_row, _row> accum_tile;  
    zero(accum_tile);

    // load from HBM to shared
    load(x_s, g.x, {0, 0, 0, 0});
    __syncthreads();

    // load from shared to register
    load(x_reg, x_s);
    __syncthreads();

    // now do the matmul and accumulate to accum_tile
    mma_ABt(accum_tile, x_reg, x_reg, accum_tile); // o = torch.matmul(x, x.transpose(1, 2))
    __syncthreads();

    // store from register to shared
    store(o_s, accum_tile);
    __syncthreads();

    // store from shared to HBM
    store(g.o, o_s, {0, 0, 0, 0});
    __syncthreads();
}


// // old code
// void dispatch_micro( bf16 *d_x, float *d_o ) {
//     using x_gl = gl<bf16, -1, -1, -1, -1, st_bf<_row, _col>>;
//     using o_gl = gl<float, -1, -1, -1, -1, st_fl<_row, _row>>;
//     using globals = micro_globals;
//     x_gl  x_arg{d_x, 1, 1, _row, _col};
//     o_gl  o_arg{d_o, 1, 1, _row, _row};
//     globals g{x_arg, o_arg};
//     unsigned long mem_size = 50480; 
//     hipFuncSetAttribute(reinterpret_cast<const void*>(
//         micro_tk),
//         hipFuncAttributeMaxDynamicSharedMemorySize,
//         mem_size
//     );
//     micro_tk<<<1,32,mem_size>>>(g);
//     hipDeviceSynchronize();
// }


void dispatch_micro(micro_globals g) {
    // MISSING??
    // I THINK WE NEED TO REDO THE LAYOUT!
    // somehow do the step? _gl  x_arg{d_x, 1, 1, _row, _col};
    // i tried to put another TK layout here but it didn't work

    unsigned long mem_size = 50480; 
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        micro_tk),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        mem_size
    );
    // micro_tk<<<1,32,mem_size>>>(g);
    micro_tk<<<g.grid(), g.block(), mem_size>>>(g);
    // micro_tk<<<1,32, mem_size>>>(g);
    hipDeviceSynchronize();
}


PYBIND11_MODULE(simple_tk, m) {
    m.doc() = "simple_tk python module";
    BIND_KERNEL(m, "micro_tk", micro_tk, micro_globals, x, o); // For wrapping kernels directly.
    BIND_FUNCTION(m, "dispatch_micro", dispatch_micro, micro_globals, x, o); // For host functions that wrap the kernel.
}
