#include "hip/hip_runtime.h"
#include "kittens.cuh"

using namespace kittens;
#define NUM_THREADS (kittens::WARP_THREADS)

#define _row 16
#define _col 32

struct micro_globals {
    using x_gl  = gl<bf16, -1, -1, -1, -1, st_bf<_row, _col>>;
    using o_gl  = gl<float, -1, -1, -1, -1, st_fl<_row, _row>>;
    x_gl x;
    o_gl o;
};

__global__ __launch_bounds__(NUM_THREADS, 1)
void micro_tk(const __grid_constant__ micro_globals g) {
    // shared memory
    extern __shared__ alignment_dummy __shm[];
    shared_allocator al((int*)&__shm[0]);
    st_bf<_row, _col> (&x_s) = al.allocate<st_bf<_row, _col>>(); // bf16 tiles
    st_fl<_row, _row> (&o_s) = al.allocate<st_fl<_row, _row>>(); // float tiles

    // register memory
    rt_bf<_row, _col> x_reg; // bf16 register
    rt_fl <_row, _row> accum_tile;  
    zero(accum_tile);

    // load from HBM to shared
    load(x_s, g.x, {0, 0, 0, 0});
    __syncthreads();

    // load from shared to register
    load(x_reg, x_s);
    __syncthreads();

    // now do the matmul and accumulate to accum_tile
    mma_ABt(accum_tile, x_reg, x_reg, accum_tile); // o = torch.matmul(x, x.transpose(1, 2))
    __syncthreads();

    // store from register to shared
    store(o_s, accum_tile);
    __syncthreads();

    // store from shared to HBM
    store(g.o, o_s, {0, 0, 0, 0});
    __syncthreads();
}

void dispatch_micro( bf16 *d_x, float *d_o ) {
    using x_gl = gl<bf16, -1, -1, -1, -1, st_bf<_row, _col>>;
    using o_gl = gl<float, -1, -1, -1, -1, st_fl<_row, _row>>;
    using globals = micro_globals;
    x_gl  x_arg{d_x, 1, 1, _row, _col};
    o_gl  o_arg{d_o, 1, 1, _row, _row};
    globals g{x_arg, o_arg};
    unsigned long mem_size = 50480; 
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        micro_tk),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        mem_size
    );
    micro_tk<<<1,32,mem_size>>>(g);
    hipDeviceSynchronize();
}
#include "harness.impl"

