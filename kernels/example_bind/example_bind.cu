#include "hip/hip_runtime.h"
#include "kittens.cuh"
#include "pyutils/pyutils.cuh"
using namespace kittens;

using my_layout = gl<float, -1, -1, -1, 64, st_fl<64,64>>; // An example layout that also instantiates a TMA descriptor on Hopper.
struct globals {
    my_layout in, out;
    dim3 grid()  { return dim3(in.batch, in.depth, in.rows); }
    dim3 block() { return dim3(in.cols); }
};
__global__ void copy_kernel(const __grid_constant__ globals g) {
    if(threadIdx.x == 0 && blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0) printf("Hello, from inside the kernel!\n");
    g.out[{blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x}] = g.in[{blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x}];
}
void run_copy_kernel(globals g) {
    printf("I am calling the kernel from the host.\n");
    copy_kernel<<<g.grid(), g.block()>>>(g);
}

PYBIND11_MODULE(example_bind, m) {
    m.doc() = "example_bind python module";
    BIND_KERNEL(m, "copy_kernel", copy_kernel, globals, in, out); // For wrapping kernels directly.
    BIND_FUNCTION(m, "wrapped_copy_kernel", run_copy_kernel, globals, in, out); // For host functions that wrap the kernel.
}
